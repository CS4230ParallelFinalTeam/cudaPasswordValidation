#include "hip/hip_runtime.h"
//#include "stdafx.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <cutil.h>
#include <unistd.h>
#include <stdbool.h>
#include <math.h>
#include <stdint.h>

// can represent 78 characters, each var will represent 13 chars
struct passNum{
uint64_t x1;
uint64_t x2;
uint64_t x3;
uint64_t x4;
uint64_t x5;
uint64_t x6;
uint64_t x7;
uint64_t x8;
uint64_t x9;
uint64_t x10;
uint64_t x11;
uint64_t x12;
uint64_t x13;
uint64_t x14;
};

extern void findPass_wrapper(passNum *passPtr, passNum *bruteNumOut, uint64_t *maxNum);
extern __global__ void findPass(uint64_t *passPtr, uint64_t *bruteNumOut, uint64_t *max);
extern void stringToNumaz(char *str, passNum *passPtr);
extern void stringToNumazAZ(char *str, passNum *passPtr);
extern void stringToNumazAZ09(char *str, passNum *passPtr);
extern void stringToNumazAZ09sc(char *str, passNum *passPtr);
extern int hipMalloc();
extern int hipMemcpy();
extern int hipFree();
extern void __syncthreads();
extern int hipMemcpyToSymbol();
__device__ bool isCracked = false;
__device__ uint64_t bruteOut = 0;

//wrapper for findPass, allocates memory then passes it to device and calls findPass
extern void findPass_wrapper(passNum *passPtr, passNum *bruteNumOut, uint64_t *maxNum)
{
	uint64_t *devI1Ptr;
	uint64_t *dev01Ptr;
	uint64_t *devI2Ptr;
	
	hipMalloc((void **)&dev01Ptr, sizeof(uint64_t));
    hipMemcpy(dev01Ptr, &bruteNumOut->x1, sizeof(uint64_t), hipMemcpyHostToDevice);
    hipMalloc((void **)&devI1Ptr, sizeof(uint64_t));
    hipMemcpy(devI1Ptr, &passPtr->x1, sizeof(uint64_t), hipMemcpyHostToDevice);
    hipMalloc((void **)&devI2Ptr, sizeof(uint64_t));
    hipMemcpy(devI2Ptr, maxNum, sizeof(uint64_t), hipMemcpyHostToDevice);
    
    dim3 dimGrid(40960, 1);
    dim3 dimBlock(512, 1);
    
    if(passPtr->x1 != 0)
    {
		findPass<<<dimGrid,dimBlock>>>(devI1Ptr, dev01Ptr, devI2Ptr);
		hipDeviceSynchronize();
		//hipMemcpy(&bruteNumOut->x1, dev01Ptr, sizeof(uint64_t), hipMemcpyDeviceToHost);
		hipMemcpyFromSymbol(&bruteNumOut->x1, HIP_SYMBOL("bruteOut"), sizeof(uint64_t),0,hipMemcpyDeviceToHost);
		bruteOut = 0;
    }
    
    if(passPtr->x2 != 0)
    {
		hipMemcpy(devI1Ptr, &passPtr->x2, sizeof(uint64_t), hipMemcpyHostToDevice);
		hipMemcpy(dev01Ptr, &bruteNumOut->x2, sizeof(uint64_t), hipMemcpyHostToDevice);
		findPass<<<dimGrid,dimBlock>>>(devI1Ptr, dev01Ptr, devI2Ptr);
		hipDeviceSynchronize();
		hipMemcpyFromSymbol(&bruteNumOut->x2, HIP_SYMBOL("bruteOut"), sizeof(uint64_t),0,hipMemcpyDeviceToHost);
		bruteOut = 0;
    }
    
        if(passPtr->x3 != 0)
    {
		hipMemcpy(devI1Ptr, &passPtr->x3, sizeof(uint64_t), hipMemcpyHostToDevice);
		hipMemcpy(dev01Ptr, &bruteNumOut->x3, sizeof(uint64_t), hipMemcpyHostToDevice);
		findPass<<<dimGrid,dimBlock>>>(devI1Ptr, dev01Ptr, devI2Ptr);
		hipDeviceSynchronize();
		hipMemcpyFromSymbol(&bruteNumOut->x3, HIP_SYMBOL("bruteOut"), sizeof(uint64_t),0,hipMemcpyDeviceToHost);
		bruteOut = 0;
    }
    
        if(passPtr->x4 != 0)
    {
		hipMemcpy(devI1Ptr, &passPtr->x4, sizeof(uint64_t), hipMemcpyHostToDevice);
		hipMemcpy(dev01Ptr, &bruteNumOut->x4, sizeof(uint64_t), hipMemcpyHostToDevice);
		findPass<<<dimGrid,dimBlock>>>(devI1Ptr, dev01Ptr, devI2Ptr);
		hipDeviceSynchronize();
		hipMemcpyFromSymbol(&bruteNumOut->x4, HIP_SYMBOL("bruteOut"), sizeof(uint64_t),0,hipMemcpyDeviceToHost);
		bruteOut = 0;
    }
    
        if(passPtr->x5 != 0)
    {
		hipMemcpy(devI1Ptr, &passPtr->x5, sizeof(uint64_t), hipMemcpyHostToDevice);
		hipMemcpy(dev01Ptr, &bruteNumOut->x5, sizeof(uint64_t), hipMemcpyHostToDevice);
		findPass<<<dimGrid,dimBlock>>>(devI1Ptr, dev01Ptr, devI2Ptr);
		hipDeviceSynchronize();
		hipMemcpyFromSymbol(&bruteNumOut->x5, HIP_SYMBOL("bruteOut"), sizeof(uint64_t),0,hipMemcpyDeviceToHost);
		bruteOut = 0;
    }
    
        if(passPtr->x6 != 0)
    {
		hipMemcpy(devI1Ptr, &passPtr->x6, sizeof(uint64_t), hipMemcpyHostToDevice);
		hipMemcpy(dev01Ptr, &bruteNumOut->x6, sizeof(uint64_t), hipMemcpyHostToDevice);
		findPass<<<dimGrid,dimBlock>>>(devI1Ptr, dev01Ptr, devI2Ptr);
		hipDeviceSynchronize();
		hipMemcpyFromSymbol(&bruteNumOut->x6, HIP_SYMBOL("bruteOut"), sizeof(uint64_t),0,hipMemcpyDeviceToHost);
		bruteOut = 0;
    }
    
        if(passPtr->x7 != 0)
    {
		hipMemcpy(devI1Ptr, &passPtr->x7, sizeof(uint64_t), hipMemcpyHostToDevice);
		hipMemcpy(dev01Ptr, &bruteNumOut->x7, sizeof(uint64_t), hipMemcpyHostToDevice);
		findPass<<<dimGrid,dimBlock>>>(devI1Ptr, dev01Ptr, devI2Ptr);
		hipDeviceSynchronize();
		hipMemcpyFromSymbol(&bruteNumOut->x7, HIP_SYMBOL("bruteOut"), sizeof(uint64_t),0,hipMemcpyDeviceToHost);
		bruteOut = 0;
    }
    
        if(passPtr->x8 != 0)
    {
		hipMemcpy(devI1Ptr, &passPtr->x8, sizeof(uint64_t), hipMemcpyHostToDevice);
		hipMemcpy(dev01Ptr, &bruteNumOut->x8, sizeof(uint64_t), hipMemcpyHostToDevice);
		findPass<<<dimGrid,dimBlock>>>(devI1Ptr, dev01Ptr, devI2Ptr);
		hipDeviceSynchronize();
		hipMemcpyFromSymbol(&bruteNumOut->x8, HIP_SYMBOL("bruteOut"), sizeof(uint64_t),0,hipMemcpyDeviceToHost);
		bruteOut = 0;
    }
    
        if(passPtr->x9 != 0)
    {
		hipMemcpy(devI1Ptr, &passPtr->x9, sizeof(uint64_t), hipMemcpyHostToDevice);
		hipMemcpy(dev01Ptr, &bruteNumOut->x9, sizeof(uint64_t), hipMemcpyHostToDevice);
		findPass<<<dimGrid,dimBlock>>>(devI1Ptr, dev01Ptr, devI2Ptr);
		hipDeviceSynchronize();
		hipMemcpyFromSymbol(&bruteNumOut->x9, HIP_SYMBOL("bruteOut"), sizeof(uint64_t),0,hipMemcpyDeviceToHost);
		bruteOut = 0;
    }
    
        if(passPtr->x10 != 0)
    {
		hipMemcpy(devI1Ptr, &passPtr->x10, sizeof(uint64_t), hipMemcpyHostToDevice);
		hipMemcpy(dev01Ptr, &bruteNumOut->x10, sizeof(uint64_t), hipMemcpyHostToDevice);
		findPass<<<dimGrid,dimBlock>>>(devI1Ptr, dev01Ptr, devI2Ptr);
		hipDeviceSynchronize();
		hipMemcpyFromSymbol(&bruteNumOut->x10, HIP_SYMBOL("bruteOut"), sizeof(uint64_t),0,hipMemcpyDeviceToHost);
		bruteOut = 0;
    }
    
        if(passPtr->x11 != 0)
    {
		hipMemcpy(devI1Ptr, &passPtr->x11, sizeof(uint64_t), hipMemcpyHostToDevice);
		hipMemcpy(dev01Ptr, &bruteNumOut->x11, sizeof(uint64_t), hipMemcpyHostToDevice);
		findPass<<<dimGrid,dimBlock>>>(devI1Ptr, dev01Ptr, devI2Ptr);
		hipDeviceSynchronize();
		hipMemcpyFromSymbol(&bruteNumOut->x11, HIP_SYMBOL("bruteOut"), sizeof(uint64_t),0,hipMemcpyDeviceToHost);
		bruteOut = 0;
    }
    
        if(passPtr->x12 != 0)
    {
		hipMemcpy(devI1Ptr, &passPtr->x12, sizeof(uint64_t), hipMemcpyHostToDevice);
		hipMemcpy(dev01Ptr, &bruteNumOut->x12, sizeof(uint64_t), hipMemcpyHostToDevice);
		findPass<<<dimGrid,dimBlock>>>(devI1Ptr, dev01Ptr, devI2Ptr);
		hipDeviceSynchronize();
		hipMemcpyFromSymbol(&bruteNumOut->x12, HIP_SYMBOL("bruteOut"), sizeof(uint64_t),0,hipMemcpyDeviceToHost);
		bruteOut = 0;
    }
    
            if(passPtr->x13 != 0)
    {
		hipMemcpy(devI1Ptr, &passPtr->x13, sizeof(uint64_t), hipMemcpyHostToDevice);
		hipMemcpy(dev01Ptr, &bruteNumOut->x13, sizeof(uint64_t), hipMemcpyHostToDevice);
		findPass<<<dimGrid,dimBlock>>>(devI1Ptr, dev01Ptr, devI2Ptr);
		hipDeviceSynchronize();
		hipMemcpyFromSymbol(&bruteNumOut->x13, HIP_SYMBOL("bruteOut"), sizeof(uint64_t),0,hipMemcpyDeviceToHost);
		bruteOut = 0;
    }
    
            if(passPtr->x14 != 0)
    {
		hipMemcpy(devI1Ptr, &passPtr->x14, sizeof(uint64_t), hipMemcpyHostToDevice);
		hipMemcpy(dev01Ptr, &bruteNumOut->x14, sizeof(uint64_t), hipMemcpyHostToDevice);
		findPass<<<dimGrid,dimBlock>>>(devI1Ptr, dev01Ptr, devI2Ptr);
		hipDeviceSynchronize();
		hipMemcpyFromSymbol(&bruteNumOut->x14, HIP_SYMBOL("bruteOut"), sizeof(uint64_t),0,hipMemcpyDeviceToHost);
		bruteOut = 0;
    }
    hipError_t cudaErr;
    cudaErr = hipGetLastError();
    printf("%s\n", hipGetErrorString(cudaErr));
    hipDeviceSynchronize();
    
    hipFree(devI1Ptr);
    hipFree(dev01Ptr);
    hipFree(devI2Ptr);
    return;
} 

extern __global__ void findPass(uint64_t *passPtr, uint64_t *bruteNumOut, uint64_t *max)
{
	//each thread gets its own passNum to increment
	uint64_t maxNum = *max;
	uint64_t bruteNum = 0;
	uint64_t passNum = *passPtr;
	int bx = blockIdx.x; //40960 blocks
	int tx = threadIdx.x; //512 threads
	int tid = tx + 512*bx;
	int stride = 40960*512;
	isCracked = false;

	__syncthreads();
	if(tid <= passNum){
		//bruteNum starts at its tid, then increments by stride
		bruteNum = tid;

		while(!isCracked)
		{
		
				//if we have found the password
				if(bruteNum >= passNum)
				{
					bruteOut = passNum;
					//*bruteNumOut = bruteNum;
					//set output to the found password
				isCracked = true; //we're done
				}

				bruteNum += stride;
		
}}
return;
}


main (int argc, char **argv) {
	//variables
	bool demoMode = 0;
	passNum pass = {0,0,0,0,0,0,0,0,0};//Users entered password
	passNum *passPtr = &pass;
	passNum brute = {0,0,0,0,0,0,0,0,0};//programs cracked password to be found
	passNum *brutePtr = &brute;
	uint64_t *maxNumaz;
	maxNumaz = (uint64_t*)malloc(sizeof(uint64_t));
	*maxNumaz = 1250000000000; //line where the gpu may timeout
	/*
	az = 				8 zs	217180147158	
	azAZ = 				6 zs	38736564342
	azAZ09 =			6 zs	168705298125
	azAZ09sc =			6 ~s	661055187436 
	
	*/
	//take in user inputs   
	char text[80];
	if(demoMode)
	char *password = getpass("Password: ");
	else
	{
		fputs("Please Enter Your Password: ", stdout);
		fflush(stdout); /* http://c-faq.com/stdio/fflush.html */
		fgets(text, sizeof text, stdin);
		printf("text = %s \n", text);
	}
	
	char *str = &text[0];
	stringToNumazAZ(str, passPtr);
	printf("x1 = %llu \n", passPtr->x1);
	printf("x2 = %llu \n", passPtr->x2);
	printf("x3 = %llu \n", passPtr->x3);
	printf("x4 = %llu \n", passPtr->x4);
	printf("x5 = %llu \n", passPtr->x5);
	printf("x6 = %llu \n", passPtr->x6);
	
	findPass_wrapper(passPtr, brutePtr, maxNumaz);
	printf("x1 = %llu \n", brutePtr->x1);
	printf("x2 = %llu \n", brutePtr->x2);
	printf("x3 = %llu \n", brutePtr->x3);
	printf("x4 = %llu \n", brutePtr->x4);
	printf("x5 = %llu \n", brutePtr->x5);
	printf("x6 = %llu \n", brutePtr->x6);
	printf("x7 = %llu \n", brutePtr->x7);
	printf("x8 = %llu \n", brutePtr->x8);
	printf("x9 = %llu \n", brutePtr->x9);
	
	return 0;
	
}

//hash function for a-z special chars(not congruent, but it doesn't have any collisions) ascii valid chars 33 - 126
void stringToNumaz(char *str, passNum *passPtr)
{

	for(int i = 0; i < 80; i++)
	{
		if(str[i] == '\n')
			return;

		
		if(i > 79)
			passPtr->x11 += (str[i] - 'a' + 1) * pow(26,i-80);
		else if(i > 71)
			passPtr->x10 += (str[i] - 'a' + 1) * pow(26,i-72);
		else if(i > 63)
			passPtr->x9 += (str[i] - 'a' + 1) * pow(26,i-64);
		else if(i > 55)
			passPtr->x8 += (str[i] - 'a' + 1) * pow(26,i-56);
		else if(i > 47)
			passPtr->x7 += (str[i] - 'a' + 1) * pow(26,i-48);
		else if(i > 39)
			passPtr->x6 += (str[i] - 'a' + 1) * pow(26,i-40);
		else if(i > 31)
			passPtr->x5 += (str[i] - 'a' + 1) * pow(26,i-32);
		else if(i > 23)
			passPtr->x4 += (str[i] - 'a' + 1) * pow(26,i-24);
		else if(i > 15)
			passPtr->x3 += (str[i] - 'a' + 1) * pow(26,i-16);
		else if(i > 7)
			passPtr->x2 += (str[i] - 'a' + 1) * pow(26,i-8);
		else
			passPtr->x1 += (str[i] - 'a' + 1) * pow(26,i);
		}
		
	return;
}

//58 possible chars, ascii A-z
void stringToNumazAZ(char *str, passNum *passPtr)
{

	for(int i = 0; i < 80; i++)
	{
		if(str[i] == '\n')
			return;


		if(i > 77)
			passPtr->x14 += (str[i] - 'A' + 1) * pow(58,i-78);
		else if(i > 71)
			passPtr->x13 += (str[i] - 'A' + 1) * pow(58,i-72);
		else if(i > 65)
			passPtr->x12 += (str[i] - 'A' + 1) * pow(58,i-66);
		else if(i > 59)
			passPtr->x11 += (str[i] - 'A' + 1) * pow(58,i-60);
		else if(i > 53)
			passPtr->x10 += (str[i] - 'A' + 1) * pow(58,i-54);
		else if(i > 47)
			passPtr->x9 += (str[i] - 'A' + 1) * pow(58,i-48);
		else if(i > 41)
			passPtr->x8 += (str[i] - 'A' + 1) * pow(58,i-42);
		else if(i > 35)
			passPtr->x7 += (str[i] - 'A' + 1) * pow(58,i-36);
		else if(i > 29)
			passPtr->x6 += (str[i] - 'A' + 1) * pow(58,i-30);
		else if(i > 23)
			passPtr->x5 += (str[i] - 'A' + 1) * pow(58,i-24);
		else if(i > 17)
			passPtr->x4 += (str[i] - 'A' + 1) * pow(58,i-18);
		else if(i > 11)
			passPtr->x3 += (str[i] - 'A' + 1) * pow(58,i-12);
		else if(i > 5)
			passPtr->x2 += (str[i] - 'A' + 1) * pow(58,i-6);
		else
			passPtr->x1 += (str[i] - 'A' + 1) * pow(58,i);
		}
		
	return;
}

//74 possible chars, asci 0 through z
void stringToNumazAZ09(char *str, passNum *passPtr)
{

	for(int i = 0; i < 80; i++)
	{
		if(str[i] == '\n')
			return;

		if(i > 77)
			passPtr->x14 += (str[i] - '0' + 1) * pow(58,i-78);
		else if(i > 71)
			passPtr->x13 += (str[i] - '0' + 1) * pow(58,i-72);
		else if(i > 65)
			passPtr->x12 += (str[i] - '0' + 1) * pow(58,i-66);
		else if(i > 59)
			passPtr->x11 += (str[i] - '0' + 1) * pow(58,i-60);
		else if(i > 53)
			passPtr->x10 += (str[i] - '0' + 1) * pow(58,i-54);
		else if(i > 47)
			passPtr->x9 += (str[i] - '0' + 1) * pow(58,i-48);
		else if(i > 41)
			passPtr->x8 += (str[i] - '0' + 1) * pow(58,i-42);
		else if(i > 35)
			passPtr->x7 += (str[i] - '0' + 1) * pow(58,i-36);
		else if(i > 29)
			passPtr->x6 += (str[i] - '0' + 1) * pow(58,i-30);
		else if(i > 23)
			passPtr->x5 += (str[i] - '0' + 1) * pow(58,i-24);
		else if(i > 17)
			passPtr->x4 += (str[i] - '0' + 1) * pow(58,i-18);
		else if(i > 11)
			passPtr->x3 += (str[i] - '0' + 1) * pow(58,i-12);
		else if(i > 5)
			passPtr->x2 += (str[i] - '0' + 1) * pow(58,i-6);
		else
			passPtr->x1 += (str[i] - '0' + 1) * pow(58,i);
		}
		
	return;
}

//74 possible chars, asci 0 through z
void stringToNumazAZ09sc(char *str, passNum *passPtr)
{

	for(int i = 0; i < 80; i++)
	{
		if(str[i] == '\n')
			return;

		if(i > 77)
			passPtr->x14 += (str[i] - '!' + 1) * pow(58,i-78);
		else if(i > 71)
			passPtr->x13 += (str[i] - '!' + 1) * pow(58,i-72);
		else if(i > 65)
			passPtr->x12 += (str[i] - '!' + 1) * pow(58,i-66);
		else if(i > 59)
			passPtr->x11 += (str[i] - '!' + 1) * pow(58,i-60);
		else if(i > 53)
			passPtr->x10 += (str[i] - '!' + 1) * pow(58,i-54);
		else if(i > 47)
			passPtr->x9 += (str[i] - '!' + 1) * pow(58,i-48);
		else if(i > 41)
			passPtr->x8 += (str[i] - '!' + 1) * pow(58,i-42);
		else if(i > 35)
			passPtr->x7 += (str[i] - '!' + 1) * pow(58,i-36);
		else if(i > 29)
			passPtr->x6 += (str[i] - '!' + 1) * pow(58,i-30);
		else if(i > 23)
			passPtr->x5 += (str[i] - '!' + 1) * pow(58,i-24);
		else if(i > 17)
			passPtr->x4 += (str[i] - '!' + 1) * pow(58,i-18);
		else if(i > 11)
			passPtr->x3 += (str[i] - '!' + 1) * pow(58,i-12);
		else if(i > 5)
			passPtr->x2 += (str[i] - '!' + 1) * pow(58,i-6);
		else
			passPtr->x1 += (str[i] - '!' + 1) * pow(58,i);
		}
		
	return;
}
