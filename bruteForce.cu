#include "hip/hip_runtime.h"
//#include "stdafx.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <cutil.h>
#include <unistd.h>
#include <stdbool.h>
#include <math.h>
#include <stdint.h>
#include <omp.h>

// can represent 78 characters, each var will represent 13 chars
struct passNum{
uint64_t x1;
uint64_t x2;
uint64_t x3;
uint64_t x4;
uint64_t x5;
uint64_t x6;
uint64_t x7;
uint64_t x8;
uint64_t x9;
uint64_t x10;
uint64_t x11;
uint64_t x12;
uint64_t x13;
uint64_t x14;
};

extern void findPass_wrapper(passNum *passPtr, passNum *bruteNumOut);
extern __global__ void findPass(uint64_t *passPtr, uint64_t *bruteNumOut);
extern void seqFindPass_wrapper(passNum *passPtr, passNum *bruteNumOut);
extern void seqFindPass(uint64_t *passPtr, uint64_t *bruteNumOut);
extern void stringToNumaz(char *str, passNum *passPtr);
extern void stringToNumazAZ(char *str, passNum *passPtr);
extern void stringToNumazAZ09(char *str, passNum *passPtr);
extern void stringToNumazAZ09sc(char *str, passNum *passPtr);
extern int hipMalloc();
extern int hipMemcpy();
extern int hipFree();
extern void __syncthreads();
extern int hipMemcpyToSymbol();
__device__ bool isCracked = false;
__device__ uint64_t bruteOut = 0;

//wrapper for findPass, allocates memory then passes it to device and calls findPass
extern void findPass_wrapper(passNum *passPtr, passNum *bruteNumOut)
{
	uint64_t *devI1Ptr;
	uint64_t *dev01Ptr;
	
	hipMalloc((void **)&dev01Ptr, sizeof(uint64_t));
    hipMemcpy(dev01Ptr, &bruteNumOut->x1, sizeof(uint64_t), hipMemcpyHostToDevice);
    hipMalloc((void **)&devI1Ptr, sizeof(uint64_t));
    hipMemcpy(devI1Ptr, &passPtr->x1, sizeof(uint64_t), hipMemcpyHostToDevice);

    
    dim3 dimGrid(40960, 1);
    dim3 dimBlock(512, 1);
    
    if(passPtr->x1 != 0)
    {
		findPass<<<dimGrid,dimBlock>>>(devI1Ptr, dev01Ptr);
		hipDeviceSynchronize();
		//hipMemcpy(&bruteNumOut->x1, dev01Ptr, sizeof(uint64_t), hipMemcpyDeviceToHost);
		hipMemcpyFromSymbol(&bruteNumOut->x1, HIP_SYMBOL("bruteOut"), sizeof(uint64_t),0,hipMemcpyDeviceToHost);
		bruteOut = 0;
    }
    
    if(passPtr->x2 != 0)
    {
		hipMemcpy(devI1Ptr, &passPtr->x2, sizeof(uint64_t), hipMemcpyHostToDevice);
		hipMemcpy(dev01Ptr, &bruteNumOut->x2, sizeof(uint64_t), hipMemcpyHostToDevice);
		findPass<<<dimGrid,dimBlock>>>(devI1Ptr, dev01Ptr);
		hipDeviceSynchronize();
		hipMemcpyFromSymbol(&bruteNumOut->x2, HIP_SYMBOL("bruteOut"), sizeof(uint64_t),0,hipMemcpyDeviceToHost);
		bruteOut = 0;
    }
    
        if(passPtr->x3 != 0)
    {
		hipMemcpy(devI1Ptr, &passPtr->x3, sizeof(uint64_t), hipMemcpyHostToDevice);
		hipMemcpy(dev01Ptr, &bruteNumOut->x3, sizeof(uint64_t), hipMemcpyHostToDevice);
		findPass<<<dimGrid,dimBlock>>>(devI1Ptr, dev01Ptr);
		hipDeviceSynchronize();
		hipMemcpyFromSymbol(&bruteNumOut->x3, HIP_SYMBOL("bruteOut"), sizeof(uint64_t),0,hipMemcpyDeviceToHost);
		bruteOut = 0;
    }
    
        if(passPtr->x4 != 0)
    {
		hipMemcpy(devI1Ptr, &passPtr->x4, sizeof(uint64_t), hipMemcpyHostToDevice);
		hipMemcpy(dev01Ptr, &bruteNumOut->x4, sizeof(uint64_t), hipMemcpyHostToDevice);
		findPass<<<dimGrid,dimBlock>>>(devI1Ptr, dev01Ptr);
		hipDeviceSynchronize();
		hipMemcpyFromSymbol(&bruteNumOut->x4, HIP_SYMBOL("bruteOut"), sizeof(uint64_t),0,hipMemcpyDeviceToHost);
		bruteOut = 0;
    }
    
        if(passPtr->x5 != 0)
    {
		hipMemcpy(devI1Ptr, &passPtr->x5, sizeof(uint64_t), hipMemcpyHostToDevice);
		hipMemcpy(dev01Ptr, &bruteNumOut->x5, sizeof(uint64_t), hipMemcpyHostToDevice);
		findPass<<<dimGrid,dimBlock>>>(devI1Ptr, dev01Ptr);
		hipDeviceSynchronize();
		hipMemcpyFromSymbol(&bruteNumOut->x5, HIP_SYMBOL("bruteOut"), sizeof(uint64_t),0,hipMemcpyDeviceToHost);
		bruteOut = 0;
    }
    
        if(passPtr->x6 != 0)
    {
		hipMemcpy(devI1Ptr, &passPtr->x6, sizeof(uint64_t), hipMemcpyHostToDevice);
		hipMemcpy(dev01Ptr, &bruteNumOut->x6, sizeof(uint64_t), hipMemcpyHostToDevice);
		findPass<<<dimGrid,dimBlock>>>(devI1Ptr, dev01Ptr);
		hipDeviceSynchronize();
		hipMemcpyFromSymbol(&bruteNumOut->x6, HIP_SYMBOL("bruteOut"), sizeof(uint64_t),0,hipMemcpyDeviceToHost);
		bruteOut = 0;
    }
    
        if(passPtr->x7 != 0)
    {
		hipMemcpy(devI1Ptr, &passPtr->x7, sizeof(uint64_t), hipMemcpyHostToDevice);
		hipMemcpy(dev01Ptr, &bruteNumOut->x7, sizeof(uint64_t), hipMemcpyHostToDevice);
		findPass<<<dimGrid,dimBlock>>>(devI1Ptr, dev01Ptr);
		hipDeviceSynchronize();
		hipMemcpyFromSymbol(&bruteNumOut->x7, HIP_SYMBOL("bruteOut"), sizeof(uint64_t),0,hipMemcpyDeviceToHost);
		bruteOut = 0;
    }
    
        if(passPtr->x8 != 0)
    {
		hipMemcpy(devI1Ptr, &passPtr->x8, sizeof(uint64_t), hipMemcpyHostToDevice);
		hipMemcpy(dev01Ptr, &bruteNumOut->x8, sizeof(uint64_t), hipMemcpyHostToDevice);
		findPass<<<dimGrid,dimBlock>>>(devI1Ptr, dev01Ptr);
		hipDeviceSynchronize();
		hipMemcpyFromSymbol(&bruteNumOut->x8, HIP_SYMBOL("bruteOut"), sizeof(uint64_t),0,hipMemcpyDeviceToHost);
		bruteOut = 0;
    }
    
        if(passPtr->x9 != 0)
    {
		hipMemcpy(devI1Ptr, &passPtr->x9, sizeof(uint64_t), hipMemcpyHostToDevice);
		hipMemcpy(dev01Ptr, &bruteNumOut->x9, sizeof(uint64_t), hipMemcpyHostToDevice);
		findPass<<<dimGrid,dimBlock>>>(devI1Ptr, dev01Ptr);
		hipDeviceSynchronize();
		hipMemcpyFromSymbol(&bruteNumOut->x9, HIP_SYMBOL("bruteOut"), sizeof(uint64_t),0,hipMemcpyDeviceToHost);
		bruteOut = 0;
    }
    
        if(passPtr->x10 != 0)
    {
		hipMemcpy(devI1Ptr, &passPtr->x10, sizeof(uint64_t), hipMemcpyHostToDevice);
		hipMemcpy(dev01Ptr, &bruteNumOut->x10, sizeof(uint64_t), hipMemcpyHostToDevice);
		findPass<<<dimGrid,dimBlock>>>(devI1Ptr, dev01Ptr);
		hipDeviceSynchronize();
		hipMemcpyFromSymbol(&bruteNumOut->x10, HIP_SYMBOL("bruteOut"), sizeof(uint64_t),0,hipMemcpyDeviceToHost);
		bruteOut = 0;
    }
    
        if(passPtr->x11 != 0)
    {
		hipMemcpy(devI1Ptr, &passPtr->x11, sizeof(uint64_t), hipMemcpyHostToDevice);
		hipMemcpy(dev01Ptr, &bruteNumOut->x11, sizeof(uint64_t), hipMemcpyHostToDevice);
		findPass<<<dimGrid,dimBlock>>>(devI1Ptr, dev01Ptr);
		hipDeviceSynchronize();
		hipMemcpyFromSymbol(&bruteNumOut->x11, HIP_SYMBOL("bruteOut"), sizeof(uint64_t),0,hipMemcpyDeviceToHost);
		bruteOut = 0;
    }
    
        if(passPtr->x12 != 0)
    {
		hipMemcpy(devI1Ptr, &passPtr->x12, sizeof(uint64_t), hipMemcpyHostToDevice);
		hipMemcpy(dev01Ptr, &bruteNumOut->x12, sizeof(uint64_t), hipMemcpyHostToDevice);
		findPass<<<dimGrid,dimBlock>>>(devI1Ptr, dev01Ptr);
		hipDeviceSynchronize();
		hipMemcpyFromSymbol(&bruteNumOut->x12, HIP_SYMBOL("bruteOut"), sizeof(uint64_t),0,hipMemcpyDeviceToHost);
		bruteOut = 0;
    }
    
            if(passPtr->x13 != 0)
    {
		hipMemcpy(devI1Ptr, &passPtr->x13, sizeof(uint64_t), hipMemcpyHostToDevice);
		hipMemcpy(dev01Ptr, &bruteNumOut->x13, sizeof(uint64_t), hipMemcpyHostToDevice);
		findPass<<<dimGrid,dimBlock>>>(devI1Ptr, dev01Ptr);
		hipDeviceSynchronize();
		hipMemcpyFromSymbol(&bruteNumOut->x13, HIP_SYMBOL("bruteOut"), sizeof(uint64_t),0,hipMemcpyDeviceToHost);
		bruteOut = 0;
    }
    
            if(passPtr->x14 != 0)
    {
		hipMemcpy(devI1Ptr, &passPtr->x14, sizeof(uint64_t), hipMemcpyHostToDevice);
		hipMemcpy(dev01Ptr, &bruteNumOut->x14, sizeof(uint64_t), hipMemcpyHostToDevice);
		findPass<<<dimGrid,dimBlock>>>(devI1Ptr, dev01Ptr);
		hipDeviceSynchronize();
		hipMemcpyFromSymbol(&bruteNumOut->x14, HIP_SYMBOL("bruteOut"), sizeof(uint64_t),0,hipMemcpyDeviceToHost);
		bruteOut = 0;
    }
    /*hipError_t cudaErr;
    cudaErr = hipGetLastError();
    printf("%s\n", hipGetErrorString(cudaErr));
    hipDeviceSynchronize();*/
    
    hipFree(devI1Ptr);
    hipFree(dev01Ptr);
    return;
} 

extern __global__ void findPass(uint64_t *passPtr, uint64_t *bruteNumOut)
{
	//each thread gets its own passNum to increment
	uint64_t bruteNum = 0;
	uint64_t passNum = *passPtr;
	int bx = blockIdx.x; //40960 blocks
	int tx = threadIdx.x; //512 threads
	int tid = tx + 512*bx;
	int stride = 40960*512;
	isCracked = false;

	__syncthreads();
	if(tid <= passNum){
		//bruteNum starts at its tid, then increments by stride
		bruteNum = tid;

		while(!isCracked)
		{
		
				//if we have found the password
				if(bruteNum >= passNum)
				{
					bruteOut = passNum;
					//*bruteNumOut = bruteNum;
					//set output to the found password
				isCracked = true; //we're done
				}

				bruteNum += stride;
		
}}
return;
}

void seqFindPass_wrapper(passNum *passPtr, passNum *bruteNumOut)
{
	if(passPtr->x1 != 0)
    {
		seqFindPass(&passPtr->x1, &bruteNumOut->x1);
    }
    
    if(passPtr->x2 != 0)
    {
		seqFindPass(&passPtr->x2, &bruteNumOut->x2);
    }
    
    if(passPtr->x3 != 0)
    {
		seqFindPass(&passPtr->x3, &bruteNumOut->x3);
    }
    
    if(passPtr->x4 != 0)
    {
		seqFindPass(&passPtr->x4, &bruteNumOut->x4);
    }
    
    if(passPtr->x5 != 0)
    {
		seqFindPass(&passPtr->x5, &bruteNumOut->x5);
    }
    
    if(passPtr->x6 != 0)
    {
		seqFindPass(&passPtr->x6, &bruteNumOut->x6);
    }
    
    if(passPtr->x7 != 0)
    {
		seqFindPass(&passPtr->x7, &bruteNumOut->x7);
    }
    
    if(passPtr->x8 != 0)
    {
		seqFindPass(&passPtr->x8, &bruteNumOut->x8);
    }
    
    if(passPtr->x9 != 0)
    {
		seqFindPass(&passPtr->x9, &bruteNumOut->x9);
    }
    
    if(passPtr->x10 != 0)
    {
		seqFindPass(&passPtr->x10, &bruteNumOut->x10);
    }
    
    if(passPtr->x11 != 0)
    {
		seqFindPass(&passPtr->x11, &bruteNumOut->x11);
    }
    
    if(passPtr->x13 != 0)
    {
		seqFindPass(&passPtr->x13, &bruteNumOut->x13);
    }
    
    if(passPtr->x14 != 0)
    {
		seqFindPass(&passPtr->x14, &bruteNumOut->x14);
    }
    
    return;
}

void seqFindPass(uint64_t *passPtr, uint64_t *bruteNumOut)
{
uint64_t bruteNum = *passPtr;
uint64_t i;
#pragma omp parallel for
for(i = 0; i <= bruteNum; i++)
{
}

*bruteNumOut = i;
return;
}


int main (int argc, char **argv) {
	//variables
	char passTypeChar[4];
	int passType = 0; //type of password
	char text[80];//users password
	passNum pass = {0,0,0,0,0,0,0,0,0,0,0,0,0,0};//Users entered password
	passNum *passPtr = &pass;
	passNum brute = {0,0,0,0,0,0,0,0,0,0,0,0,0,0};//programs cracked password to be found
	passNum *brutePtr = &brute;
	passNum seqbrute = {0,0,0,0,0,0,0,0,0,0,0,0,0,0};//seq programs cracked password to be found
	passNum *seqbrutePtr = &seqbrute;
	
	//take in user inputs   
	printf("\n1: a-z\n2: a-z A-Z\n3: a-z A-Z 0-9\n4: a-z A-Z 0-9 Special Characters\n");
	printf("Please Enter The Type of Password: ");
	fflush(stdout); /* http://c-faq.com/stdio/fflush.html */
	fgets(passTypeChar, sizeof passTypeChar, stdin);
	passType = atoi(passTypeChar);
	fputs("Please Enter Your Password: ", stdout);
	fflush(stdout); /* http://c-faq.com/stdio/fflush.html */
	fgets(text, sizeof text, stdin);
	
	char *str = &text[0];
	switch(passType)
	{
	case 1:
	stringToNumaz(str, passPtr);
	break;
	
	case 2:
	stringToNumazAZ(str, passPtr);
	break;
	
	case 3:
	stringToNumazAZ09(str, passPtr);
	break;
	
	case 4:
	stringToNumazAZ09sc(str, passPtr);
	break;
	
	default:
	printf("Please enter a valid password type!\n");
	return 0;
	}
	
	printf("Working...\n");
	
	// create CUDA event handles for timing purposes
	hipEvent_t start_event, stop_event;
	float elapsed_time_gpu, elapsed_time_cpu;
  
	CUDA_SAFE_CALL( hipEventCreate(&start_event) );
	CUDA_SAFE_CALL( hipEventCreate(&stop_event) );
	hipEventRecord(start_event, 0);   
	seqFindPass_wrapper(passPtr, seqbrutePtr);
	hipEventRecord(stop_event, 0);
	hipEventSynchronize(stop_event);
	CUDA_SAFE_CALL( hipEventElapsedTime(&elapsed_time_cpu,start_event, stop_event) )
  
	CUDA_SAFE_CALL( hipEventCreate(&start_event) );
	CUDA_SAFE_CALL( hipEventCreate(&stop_event) );
	hipEventRecord(start_event, 0);   
	findPass_wrapper(passPtr, brutePtr);
	hipDeviceSynchronize();
	hipEventRecord(stop_event, 0);
	hipEventSynchronize(stop_event);
	CUDA_SAFE_CALL( hipEventElapsedTime(&elapsed_time_gpu,start_event, stop_event) )
  
  //print out statistics
	printf("Found!\nCPU Brute Force Time: %.2f msec\n", elapsed_time_cpu);
	printf("GPU Brute Force Time: %.2f msec\n", elapsed_time_gpu);
	printf("GPU Speedup: %.2f\n", elapsed_time_cpu/elapsed_time_gpu);
	printf("Your password is %s\n",text);
	printf("The computation tried the following number of strings:\n");
	
	printf("%llu +\n", brutePtr->x1);
	if(brutePtr->x2 > 0)
		printf("%llu +\n", brutePtr->x2);
	if(brutePtr->x3 > 0)
		printf("%llu +\n", brutePtr->x3);
	if(brutePtr->x4 > 0)
		printf("%llu +\n", brutePtr->x4);
	if(brutePtr->x5 > 0)
		printf("%llu +\n", brutePtr->x5);
	if(brutePtr->x6 > 0)
		printf("%llu +\n", brutePtr->x6);
	if(brutePtr->x7 > 0)
		printf("%llu +\n", brutePtr->x7);
	if(brutePtr->x8 > 0)
		printf("%llu +\n", brutePtr->x8);
	if(brutePtr->x9 > 0)
		printf("%llu +\n", brutePtr->x9);
	if(brutePtr->x10 > 0)
		printf("%llu +\n", brutePtr->x10);
	if(brutePtr->x11 > 0)
		printf("%llu +\n", brutePtr->x11);
	if(brutePtr->x12 > 0)
		printf("%llu +\n", brutePtr->x12);
	if(brutePtr->x13 > 0)
		printf("%llu +\n", brutePtr->x13);
	if(brutePtr->x14 > 0)
		printf("%llu +\n", brutePtr->x14);
	
	return 0;
}

//hash function for a-z special chars(not congruent, but it doesn't have any collisions) ascii valid chars 33 - 126
void stringToNumaz(char *str, passNum *passPtr)
{
	
	for(int i = 0; i < 80; i++)
	{
		if(str[i] == '\n')
			return;
		
		if(i > 79)
			passPtr->x11 += (str[i] - 'a' + 1) * pow(26,i-80);
		else if(i > 71)
			passPtr->x10 += (str[i] - 'a' + 1) * pow(26,i-72);
		else if(i > 63)
			passPtr->x9 += (str[i] - 'a' + 1) * pow(26,i-64);
		else if(i > 55)
			passPtr->x8 += (str[i] - 'a' + 1) * pow(26,i-56);
		else if(i > 47)
			passPtr->x7 += (str[i] - 'a' + 1) * pow(26,i-48);
		else if(i > 39)
			passPtr->x6 += (str[i] - 'a' + 1) * pow(26,i-40);
		else if(i > 31)
			passPtr->x5 += (str[i] - 'a' + 1) * pow(26,i-32);
		else if(i > 23)
			passPtr->x4 += (str[i] - 'a' + 1) * pow(26,i-24);
		else if(i > 15)
			passPtr->x3 += (str[i] - 'a' + 1) * pow(26,i-16);
		else if(i > 7)
			passPtr->x2 += (str[i] - 'a' + 1) * pow(26,i-8);
		else
			passPtr->x1 += (str[i] - 'a' + 1) * pow(26,i);
		}
		
	return;
}

//58 possible chars, ascii A-z
void stringToNumazAZ(char *str, passNum *passPtr)
{

	for(int i = 0; i < 80; i++)
	{
		if(str[i] == '\n')
			return;


		if(i > 77)
			passPtr->x14 += (str[i] - 'A' + 1) * pow(58,i-78);
		else if(i > 71)
			passPtr->x13 += (str[i] - 'A' + 1) * pow(58,i-72);
		else if(i > 65)
			passPtr->x12 += (str[i] - 'A' + 1) * pow(58,i-66);
		else if(i > 59)
			passPtr->x11 += (str[i] - 'A' + 1) * pow(58,i-60);
		else if(i > 53)
			passPtr->x10 += (str[i] - 'A' + 1) * pow(58,i-54);
		else if(i > 47)
			passPtr->x9 += (str[i] - 'A' + 1) * pow(58,i-48);
		else if(i > 41)
			passPtr->x8 += (str[i] - 'A' + 1) * pow(58,i-42);
		else if(i > 35)
			passPtr->x7 += (str[i] - 'A' + 1) * pow(58,i-36);
		else if(i > 29)
			passPtr->x6 += (str[i] - 'A' + 1) * pow(58,i-30);
		else if(i > 23)
			passPtr->x5 += (str[i] - 'A' + 1) * pow(58,i-24);
		else if(i > 17)
			passPtr->x4 += (str[i] - 'A' + 1) * pow(58,i-18);
		else if(i > 11)
			passPtr->x3 += (str[i] - 'A' + 1) * pow(58,i-12);
		else if(i > 5)
			passPtr->x2 += (str[i] - 'A' + 1) * pow(58,i-6);
		else
			passPtr->x1 += (str[i] - 'A' + 1) * pow(58,i);
		}
		
	return;
}

//74 possible chars, asci 0 through z
void stringToNumazAZ09(char *str, passNum *passPtr)
{

	for(int i = 0; i < 80; i++)
	{
		if(str[i] == '\n')
			return;

		if(i > 77)
			passPtr->x14 += (str[i] - '0' + 1) * pow(58,i-78);
		else if(i > 71)
			passPtr->x13 += (str[i] - '0' + 1) * pow(58,i-72);
		else if(i > 65)
			passPtr->x12 += (str[i] - '0' + 1) * pow(58,i-66);
		else if(i > 59)
			passPtr->x11 += (str[i] - '0' + 1) * pow(58,i-60);
		else if(i > 53)
			passPtr->x10 += (str[i] - '0' + 1) * pow(58,i-54);
		else if(i > 47)
			passPtr->x9 += (str[i] - '0' + 1) * pow(58,i-48);
		else if(i > 41)
			passPtr->x8 += (str[i] - '0' + 1) * pow(58,i-42);
		else if(i > 35)
			passPtr->x7 += (str[i] - '0' + 1) * pow(58,i-36);
		else if(i > 29)
			passPtr->x6 += (str[i] - '0' + 1) * pow(58,i-30);
		else if(i > 23)
			passPtr->x5 += (str[i] - '0' + 1) * pow(58,i-24);
		else if(i > 17)
			passPtr->x4 += (str[i] - '0' + 1) * pow(58,i-18);
		else if(i > 11)
			passPtr->x3 += (str[i] - '0' + 1) * pow(58,i-12);
		else if(i > 5)
			passPtr->x2 += (str[i] - '0' + 1) * pow(58,i-6);
		else
			passPtr->x1 += (str[i] - '0' + 1) * pow(58,i);
		}
		
	return;
}

//74 possible chars, asci 0 through z
void stringToNumazAZ09sc(char *str, passNum *passPtr)
{

	for(int i = 0; i < 80; i++)
	{
		if(str[i] == '\n')
			return;

		if(i > 77)
			passPtr->x14 += (str[i] - '!' + 1) * pow(58,i-78);
		else if(i > 71)
			passPtr->x13 += (str[i] - '!' + 1) * pow(58,i-72);
		else if(i > 65)
			passPtr->x12 += (str[i] - '!' + 1) * pow(58,i-66);
		else if(i > 59)
			passPtr->x11 += (str[i] - '!' + 1) * pow(58,i-60);
		else if(i > 53)
			passPtr->x10 += (str[i] - '!' + 1) * pow(58,i-54);
		else if(i > 47)
			passPtr->x9 += (str[i] - '!' + 1) * pow(58,i-48);
		else if(i > 41)
			passPtr->x8 += (str[i] - '!' + 1) * pow(58,i-42);
		else if(i > 35)
			passPtr->x7 += (str[i] - '!' + 1) * pow(58,i-36);
		else if(i > 29)
			passPtr->x6 += (str[i] - '!' + 1) * pow(58,i-30);
		else if(i > 23)
			passPtr->x5 += (str[i] - '!' + 1) * pow(58,i-24);
		else if(i > 17)
			passPtr->x4 += (str[i] - '!' + 1) * pow(58,i-18);
		else if(i > 11)
			passPtr->x3 += (str[i] - '!' + 1) * pow(58,i-12);
		else if(i > 5)
			passPtr->x2 += (str[i] - '!' + 1) * pow(58,i-6);
		else
			passPtr->x1 += (str[i] - '!' + 1) * pow(58,i);
		}
		
	return;
}
