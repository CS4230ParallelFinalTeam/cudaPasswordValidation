#include "hip/hip_runtime.h"
//#include "stdafx.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <cutil.h>
#include <unistd.h>
#include <stdbool.h>
#include <math.h>
#include <stdint.h>

// can represent 78 characters, each var will represent 13 chars
struct passNum{
uint64_t x1;
uint64_t x2;
uint64_t x3;
uint64_t x4;
uint64_t x5;
uint64_t x6;
uint64_t x7;
uint64_t x8;
uint64_t x9;
};

extern void findPass_wrapper(passNum *passPtr, passNum *bruteNumOut, uint64_t *maxNum);
extern __global__ void findPass(passNum *passPtr, passNum *bruteNumOut, uint64_t *max);
extern void stringToNumaz(char *str, passNum *passPtr);
extern void stringToNumazAZ(char *str, passNum *passPtr);
extern void stringToNumazAZ09(char *str, passNum *passPtr);
extern void stringToNumazAZ09sc(char *str, passNum *passPtr);
extern int hipMalloc();
extern int hipMemcpy();
extern int hipFree();
extern void __syncthreads();
extern int hipMemcpyToSymbol();
__device__ bool isCracked = false;

//wrapper for findPass, allocates memory then passes it to device and calls findPass
extern void findPass_wrapper(passNum *passPtr, passNum *bruteNumOut, uint64_t *maxNum)
{
	passNum *devI1Ptr;
	passNum *dev01Ptr;
	uint64_t *devI2Ptr;
	hipMalloc((void **)&dev01Ptr, sizeof(passNum));
    hipMemcpy(dev01Ptr, bruteNumOut, sizeof(passNum), hipMemcpyHostToDevice);
    hipMalloc((void **)&devI1Ptr, sizeof(passNum));
    hipMemcpy(devI1Ptr, passPtr, sizeof(passNum), hipMemcpyHostToDevice);
    hipMalloc((void **)&devI2Ptr, sizeof(int));
    hipMemcpy(devI2Ptr, maxNum, sizeof(uint64_t), hipMemcpyHostToDevice);
    
    dim3 dimGrid(43, 1);
    dim3 dimBlock(96, 1);
    findPass<<<dimGrid,dimBlock>>>(devI1Ptr, dev01Ptr, devI2Ptr);
    hipMemcpy(bruteNumOut, dev01Ptr, sizeof(passNum), hipMemcpyDeviceToHost);
    hipError_t cudaErr;
    cudaErr = hipGetLastError();
    printf("%s\n", hipGetErrorString(cudaErr));
    hipDeviceSynchronize();
    hipFree(devI1Ptr);
    hipFree(dev01Ptr);
    hipFree(devI2Ptr);
    return;
} 

extern __global__ void findPass(passNum *passPtr, passNum *bruteNumOut, uint64_t *max)
{
	//each thread gets its own passNum to increment
	uint64_t maxNum = *max;
	passNum bruteNum = {0,0,0,0,0,0,0,0,333};
	int bx = blockIdx.x;
	int tx = threadIdx.x;
	int tid = tx + 96*bx;
	int stride = 96*43;
	bruteNumOut->x9 = 456;
	
	//bruteNum starts at its tid, then increments by stride
	bruteNum.x1 = tid;
	bruteNum.x2 = tid;
	bruteNum.x3 = tid;
	bruteNum.x4 = tid;
	bruteNum.x5 = tid;
	bruteNum.x6 = tid;
	bruteNum.x7 = tid;
	bruteNum.x8 = tid;
	bruteNum.x9 = tid;
	
	while(!isCracked)
	{
	/*
		if(bruteNum.x8 >= maxNum){
			bruteNum.x9 += stride;
			bruteNum.x8 = maxNum;}
		else if(bruteNum.x7 >= maxNum){
			bruteNum.x8 += stride;
			bruteNum.x7 = maxNum;}
		else if(bruteNum.x6 >= maxNum){
			bruteNum.x7 += stride;
			bruteNum.x6 = maxNum;}
		else if(bruteNum.x5 >= maxNum){
			bruteNum.x6 += stride;
			bruteNum.x5 = maxNum;}
		else if(bruteNum.x4 >= maxNum){
			bruteNum.x5 += stride;
			bruteNum.x4 = maxNum;}
		else if(bruteNum.x3 >= maxNum){
			bruteNum.x4 += stride;
			bruteNum.x3 = maxNum;}
		else if(bruteNum.x2 >= maxNum){
			bruteNum.x3 += stride;
			bruteNum.x2 = maxNum;}
		else if(bruteNum.x1 >= maxNum){
			bruteNum.x2 += stride;
			bruteNum.x1 = maxNum;}
		else
			bruteNum.x1 += stride;
			*/
			
			bruteNum.x1 += stride;
			if(bruteNum.x1 >= 500000)
			{
			*bruteNumOut = bruteNum;
			bruteNumOut->x6 = maxNum;
			isCracked = true;
			}
			//if we have found the password
			if((bruteNum.x1 == passPtr->x1) &&
			(bruteNum.x2 == passPtr->x2) &&
			(bruteNum.x3 == passPtr->x3) &&
			(bruteNum.x4 == passPtr->x4) &&
			(bruteNum.x5 == passPtr->x5) &&
			(bruteNum.x6 == passPtr->x6) &&
			(bruteNum.x7 == passPtr->x7) &&
			(bruteNum.x8 == passPtr->x8) &&
			(bruteNum.x9 == passPtr->x9))
			{
			isCracked = true; //we're done
			*bruteNumOut = bruteNum; //set output to the found password
			}
	}
	
	__syncthreads();
	return;
}


main (int argc, char **argv) {
	//variables
	bool demoMode = 0;
	passNum pass = {0,0,0,0,0,0,0,0,111};//Users entered password
	passNum *passPtr = &pass;
	passNum brute = {0,0,0,0,0,0,0,0,222};//programs cracked password to be found
	passNum *brutePtr = &brute;
	uint64_t *maxNumaz;
	maxNumaz = (uint64_t*)malloc(sizeof(uint64_t));
	*maxNumaz = 2580398988131886080;
	//take in user inputs   
	char text[80];
	if(demoMode)
	char *password = getpass("Password: ");
	else
	{
		fputs("Please Enter Your Password: ", stdout);
		fflush(stdout); /* http://c-faq.com/stdio/fflush.html */
		fgets(text, sizeof text, stdin);
		printf("text = %s \n", text);
	}
	
	char *str = &text[0];
	stringToNumaz(str, passPtr);
	printf("x1 = %llu \n", passPtr->x1);
	printf("x2 = %llu \n", passPtr->x2);
	printf("x3 = %llu \n", passPtr->x3);
	printf("x4 = %llu \n", passPtr->x4);
	printf("x5 = %llu \n", passPtr->x5);
	printf("x6 = %llu \n", passPtr->x6);
	
	findPass_wrapper(passPtr, brutePtr, maxNumaz);
	printf("x1 = %llu \n", brutePtr->x1);
	printf("x2 = %llu \n", brutePtr->x2);
	printf("x3 = %llu \n", brutePtr->x3);
	printf("x4 = %llu \n", brutePtr->x4);
	printf("x5 = %llu \n", brutePtr->x5);
	printf("x6 = %llu \n", brutePtr->x6);
	printf("x7 = %llu \n", brutePtr->x7);
	printf("x8 = %llu \n", brutePtr->x8);
	printf("x9 = %llu \n", brutePtr->x9);
	
	return 0;
	
}

//hash function for a-z special chars(not congruent, but it doesn't have any collisions) ascii valid chars 33 - 126
void stringToNumaz(char *str, passNum *passPtr)
{

	for(int i = 0; i < 80; i++)
	{
		if(str[i] == '\n')
			return;

		
		if(i > 64)
			passPtr->x6 += (str[i] - 'a' + 1) * pow(26,i-65);
		else if(i > 51)
			passPtr->x5 += (str[i] - 'a' + 1) * pow(26,i-51);
		else if(i > 38)
			passPtr->x4 += (str[i] - 'a' + 1) * pow(26,i-39);
		else if(i > 25)
			passPtr->x3 += (str[i] - 'a' + 1) * pow(26,i-26);
		else if(i > 12)
			passPtr->x2 += (str[i] - 'a' + 1) * pow(26,i-13);
		else
			passPtr->x1 += (str[i] - 'a' + 1) * pow(26,i);
		}
		
	return;
}

//58 possible chars, ascii A-z
void stringToNumazAZ(char *str, passNum *passPtr)
{

	for(int i = 0; i < 80; i++)
	{
		if(str[i] == '\n')
			return;

		if(i > 79)
			passPtr->x9 += (str[i] - 'A' + 1) * pow(58,i-80);
		else if(i > 69)
			passPtr->x8 += (str[i] - 'A' + 1) * pow(58,i-70);
		else if(i > 59)
			passPtr->x7 += (str[i] - 'A' + 1) * pow(58,i-60);
		else if(i > 49)
			passPtr->x6 += (str[i] - 'A' + 1) * pow(58,i-50);
		else if(i > 39)
			passPtr->x5 += (str[i] - 'A' + 1) * pow(58,i-40);
		else if(i > 29)
			passPtr->x4 += (str[i] - 'A' + 1) * pow(58,i-30);
		else if(i > 19)
			passPtr->x3 += (str[i] - 'A' + 1) * pow(58,i-20);
		else if(i > 9)
			passPtr->x2 += (str[i] - 'A' + 1) * pow(58,i-10);
		else
			passPtr->x1 += (str[i] - 'A' + 1) * pow(58,i);
		}
		
	return;
}

//74 possible chars, asci 0 through z
void stringToNumazAZ09(char *str, passNum *passPtr)
{

	for(int i = 0; i < 80; i++)
	{
		if(str[i] == '\n')
			return;

		if(i > 79)
			passPtr->x9 += (str[i] - '0' + 1) * pow(74,i-80);
		else if(i > 69)
			passPtr->x8 += (str[i] - '0' + 1) * pow(74,i-70);
		else if(i > 59)
			passPtr->x7 += (str[i] - '0' + 1) * pow(74,i-60);
		else if(i > 49)
			passPtr->x6 += (str[i] - '0' + 1) * pow(74,i-50);
		else if(i > 39)
			passPtr->x5 += (str[i] - '0' + 1) * pow(74,i-40);
		else if(i > 29)
			passPtr->x4 += (str[i] - '0' + 1) * pow(74,i-30);
		else if(i > 19)
			passPtr->x3 += (str[i] - '0' + 1) * pow(74,i-20);
		else if(i > 9)
			passPtr->x2 += (str[i] - '0' + 1) * pow(74,i-10);
		else 
			passPtr->x1 += (str[i] - '0' + 1) * pow(74,i);
		}
		
	return;
}

//74 possible chars, asci 0 through z
void stringToNumazAZ09sc(char *str, passNum *passPtr)
{

	for(int i = 0; i < 80; i++)
	{
		if(str[i] == '\n')
			return;

		if(i > 71)
			passPtr->x9 += (str[i] - '!' + 1) * pow(93,i-72);
		else if(i > 62)
			passPtr->x8 += (str[i] - '!' + 1) * pow(93,i-63);
		else if(i > 53)
			passPtr->x7 += (str[i] - '!' + 1) * pow(93,i-54);
		else if(i > 44)
			passPtr->x6 += (str[i] - '!' + 1) * pow(93,i-45);
		else if(i > 35)
			passPtr->x5 += (str[i] - '!' + 1) * pow(93,i-36);
		else if(i > 26)
			passPtr->x4 += (str[i] - '!' + 1) * pow(93,i-27);
		else if(i > 17)
			passPtr->x3 += (str[i] - '!' + 1) * pow(93,i-18);
		else if(i > 8)
			passPtr->x2 += (str[i] - '!' + 1) * pow(93,i-9);
		else
			passPtr->x1 += (str[i] - '!' + 1) * pow(93,i);
		}
		
	return;
}
